#include "hip/hip_runtime.h"
#include "dotprod.h"

#include <cassert>
#include <hipcub/hipcub.hpp>

namespace {
// Not a very smart implementation!

template <typename T>
__global__ void elemwise_product(T* res, T const* __restrict__ a,
                                 T const* __restrict__ b, size_t size) {
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    res[index] = a[index] * b[index];
  }
}

// TODO move allocation to static variables to make it fast

template <typename T>
T dotprod_impl(std::vector<T> const& a, std::vector<T> const& b) {
  assert(a.size() == b.size());

  T* dev_a;
  // TODO error checking
  size_t memsize = sizeof(T) * a.size();
  hipMalloc(&dev_a, memsize);
  hipMemcpy(dev_a, a.data(), memsize, hipMemcpyHostToDevice);
  T* dev_b;
  hipMalloc(&dev_b, memsize);
  hipMemcpy(dev_b, b.data(), memsize, hipMemcpyHostToDevice);

  T* res_elemwise_product;
  hipMalloc(&res_elemwise_product, memsize);

  int threads_per_block = 512;
  int n_blocks = (a.size() - 1) / threads_per_block + 1;
  elemwise_product<<<n_blocks, threads_per_block>>>(res_elemwise_product, dev_a,
                                                    dev_b, a.size());

  T* dev_result;
  hipMalloc(&dev_result, sizeof(T));

  void* d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;

  hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                         res_elemwise_product, dev_result, a.size());

  hipMalloc(&d_temp_storage, temp_storage_bytes);

  hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                         res_elemwise_product, dev_result, a.size());

  T result;
  hipMemcpy(&result, dev_result, sizeof(T), hipMemcpyDeviceToHost);

  // TODO free
  return result;
}
}  // namespace

double dotprod(std::vector<double> const& a, std::vector<double> const& b) {
  return dotprod_impl(a, b);
}

float dotprod(std::vector<float> const& a, std::vector<float> const& b) {
  return dotprod_impl(a, b);
}
